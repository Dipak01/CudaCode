#include "hip/hip_runtime.h"
/*
INPUT: An integer n and an array A = (a_1, ..., a_n) of floating point numbers .

OUTPUT : An array of legth n : (a_1, 2 * a_1 + a_2, 3 * a_1 + 2 * a_2 + a_3, 4 * a_1 + 3 * a_2 + 2 * a_3 + a_4, ...)
		 That is, the ith member of your array must be i*a_1 + (i - 1)*a_2 + ... + 2 * a_{ i - 1 } +a_i.

Design a parallel algorithm that runs in TIME(log n) and implement it on the CUDA platform.

Example input : (2, 0, 7)
Example output : (2, 4, 13)
*/

#include <iostream>
#include <time.h>
#include "hip/hip_runtime.h"
#include ""

#define THREAD 1024
#define POWER 25

using namespace std;

// Kernel function to add the elements of two arrays
__global__
void add(int n, float *x, float *y)
{
	//blockID*blockDim + threadID
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	//threads are 16 but vectors are only 15. So to handle this extra thread we added this check.
	if (i < n) {
		for (int j = i; j >= 0; j--) {
			y[i] = y[i] + (j + 1) * x[i - j];
		}
	}
}

int main(void)
{
	int n = 1024; //1 << POWER; //left shift or 2^Power

	float *x, *y, *d_x, *d_y;

	//Allocate memory on CPU
	x = (float*)malloc(n * sizeof(float));
	y = (float*)malloc(n * sizeof(float));

	//Allocate memory on GPU
	hipMalloc(&d_x, n * sizeof(float));
	hipMalloc(&d_y, n * sizeof(float));

	// initialize x and y arrays on the host
	for (int i = 0; i < n; i++) {
		x[i] = 0.0f + i;
		y[i] = 0.0f;
	}

	//Copy memory from CPU to GPU 
	//destination, source, size of mem we need to copy, direction
	hipMemcpy(d_x, x, n * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_y, y, n * sizeof(float), hipMemcpyHostToDevice);

	//Sequential add vectors
	clock_t begin = clock();
	for (int i = 0; i < n; i++) {
		for (int j = i; j >= 0; j--) {
	//		y[i] = y[i] + (j + 1) * x[i - j];
		}		
	}
	clock_t end = clock();
	double time_spent = (double)(end - begin) / CLOCKS_PER_SEC * 1000;
	cout << "The running time for sequential addtition is " << time_spent << " miliseconds." << endl;


	begin = clock();

	// Perform Addition on GPU
	//ceiling done as default there is floor and 1 block of threads will be ignored.
	//Alternate thing: n/thread + (n % thread != 0)
	add << <(n + THREAD - 1) / THREAD, THREAD >> >(n, d_x, d_y);

	end = clock();
	time_spent = (double)(end - begin) / CLOCKS_PER_SEC * 1000;
	cout << "The running time for parallel addition is ";
	cout << time_spent << " miliseconds." << endl;


	//Copy memory from GPU to CPU 
	hipMemcpy(y, d_y, n * sizeof(float), hipMemcpyDeviceToHost);


	bool flag = true;
	//Verify Results
	for (int i = 0; i < n; i++){
		cout << y[i] <<", ";
	}
	//if (flag) cout << "Correct! Welcome to CUDA world!" << endl;

	//Free memory on GPU
	hipFree(d_x);
	hipFree(d_y);

	//Free memory on CPU
	free(x);
	free(y);

	return 0;
}