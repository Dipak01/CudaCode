#include "hip/hip_runtime.h"
//Explain the odd behavior of the following puzzle program 
//1. run it several times; 
//2. change the numbers 256, 256 to other pairs of integers

#include <iostream>
#include <stdio.h>
#include "hip/hip_runtime.h"
#include ""

using namespace std;

// Kernel function to increase a counter

__global__   
void count(int *x) { 
	*x = *x + 1; 
}

int main(void){

	int  *x, *d_x;

	//Allocate memory on CPU

	x = (int*)malloc(sizeof(int));

	//Allocate memory on GPU

	hipMalloc(&d_x, sizeof(int));

	// initialize x on the host

	*x = 0;

	//Copy memory from CPU to GPU 

	hipMemcpy(d_x, x, sizeof(int), hipMemcpyHostToDevice);

	// Perform Counting on GPU

	count << < 16, 128 >> > (d_x);

	//Copy memory from GPU to CPU

	hipMemcpy(x, d_x, sizeof(int), hipMemcpyDeviceToHost);

	//Print Results

	printf("x is %d\n", *x);



	//Free memory on GPU
	hipFree(d_x);
	//Free memory on CPU       
	free(x);

	return 0;

}